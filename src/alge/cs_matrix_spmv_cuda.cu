#include "hip/hip_runtime.h"
/*============================================================================
 * Sparse Matrix-vector multiplication kernels using CUDA.
 *============================================================================*/

/*
  This file is part of code_saturne, a general-purpose CFD tool.

  Copyright (C) 1998-2024 EDF S.A.

  This program is free software; you can redistribute it and/or modify it under
  the terms of the GNU General Public License as published by the Free Software
  Foundation; either version 2 of the License, or (at your option) any later
  version.

  This program is distributed in the hope that it will be useful, but WITHOUT
  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
  FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
  details.

  You should have received a copy of the GNU General Public License along with
  this program; if not, write to the Free Software Foundation, Inc., 51 Franklin
  Street, Fifth Floor, Boston, MA 02110-1301, USA.
*/

/*----------------------------------------------------------------------------*/

#include "cs_defs.h"

/*----------------------------------------------------------------------------
 * Standard library headers
 *----------------------------------------------------------------------------*/

#include <stdarg.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <math.h>

#if defined(HAVE_MPI)
#include <mpi.h>
#endif

#if defined(HAVE_CUSPARSE)
#include <hipsparse.h>
#endif

/*----------------------------------------------------------------------------
 * Local headers
 *----------------------------------------------------------------------------*/

#include "bft_mem.h"
#include "bft_error.h"
#include "bft_printf.h"

#include "cs_base.h"
#include "cs_base_cuda.h"
#include "cs_cuda_contrib.h"
#include "cs_halo.h"
#include "cs_halo_perio.h"
#include "cs_log.h"
#include "cs_timer.h"

/*----------------------------------------------------------------------------
 *  Header for the current file
 *----------------------------------------------------------------------------*/

#include "cs_matrix.h"
#include "cs_matrix_priv.h"
#include "cs_matrix_spmv.h"

/*----------------------------------------------------------------------------*/
/*! \file cs_matrix_spmv_cuda.c
 *
 * \brief Sparse Matrix SpMV operations with CUDA.
 */
/*----------------------------------------------------------------------------*/

/*! \cond DOXYGEN_SHOULD_SKIP_THIS */

/*============================================================================
 * Local macro definitions
 *============================================================================*/

/*----------------------------------------------------------------------------
 * Compatibility macro for __ldg (load from generic memory) intrinsic,
 * forcing load from read-only texture cache.
 *
 * This was not available in (very old) CUDA architectures.
 *----------------------------------------------------------------------------*/

#if __CUDA_ARCH__ < 350
#define __ldg(ptr) *(ptr);
#endif

/*=============================================================================
 * Local Type Definitions
 *============================================================================*/

#if defined(HAVE_CUSPARSE)

/* Mapping of matrix coefficients and structure to cuSPARSE */
/*----------------------------------------------------------*/

typedef struct _cs_matrix_cusparse_map_t {

#if defined(HAVE_CUSPARSE_GENERIC_API)

  bool  block_diag;             /* Use identity blocks diagonal structure ? */

  hipsparseSpMatDescr_t  matA;   /* Handle to cusparse Matrix */

  hipsparseDnMatDescr_t  matX;   /* Handle to cusparse Matrix (blocked vector) */
  hipsparseDnMatDescr_t  matY;   /* Handle to cusparse Matrix (blocked vector) */

  hipsparseDnVecDescr_t  vecX;   /* Handle to cusparse Vector */
  hipsparseDnVecDescr_t  vecY;   /* Handle to cusparse output Vector */

  void  *vecXValues;            /* Pointer to vector values */
  void  *vecYValues;            /* Pointer to vector values */

  void  *dBuffer;               /* Associated buffer */

#endif

  /* When not using generic API */

  int  nnz;                   /* Number of nonzeroes */
  cusparseMatDescr  *descrA;  /* Handle to cusparse Matrix description */

  void  *d_row_index;         /* Pointer to row index */
  void  *d_col_id;            /* Pointer to column ids */
  void  *d_e_val;             /* Pointer to matrix extradiagonal values */

} cs_matrix_cusparse_map_t;

#endif // defined(HAVE_CUSPARSE)

/*============================================================================
 *  Global variables
 *============================================================================*/

static hipStream_t _stream = 0;

#if defined(HAVE_CUSPARSE)

static hipsparseHandle_t  _handle = NULL;

#endif

/*============================================================================
 * Private function definitions
 *============================================================================*/

/*----------------------------------------------------------------------------*/
/* \brief Zero range of elements.
 *
 * \param[in]   n   number of elements
 * \param[out]  x   resulting vector
 */
/*----------------------------------------------------------------------------*/

__global__ static void
_zero_range(cs_lnum_t    n,
            cs_real_t   *__restrict__ x)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;

  if (ii < n)
    x[ii] = 0;
}

/*----------------------------------------------------------------------------*/
/* \brief Local diagonal contribution y = Da.x  + y.
 *
 * \param[in]   n_rows      number of local rows
 * \param[in]   n_cols_ext  number of local columns (with ghosts)
 * \param[in]   d_val       pointer to diagonal matrix values
 * \param[in]   x           multipliying vector values
 * \param[out]  y           resulting vector
 */
/*----------------------------------------------------------------------------*/

__global__ static void
_mat_vect_p_l_native_diag(cs_lnum_t         n_rows,
                          cs_lnum_t         n_cols_ext,
                          const cs_real_t  *__restrict__ d_val,
                          const cs_real_t  *__restrict__ x,
                          cs_real_t        *__restrict__ y)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;

  if (ii < n_rows)
    y[ii] = d_val[ii] * x[ii];

  else if (ii < n_cols_ext)
    y[ii] = 0;
}

/*----------------------------------------------------------------------------
 * SpMV extradiagonal terms using native to face-based array and scatter
 * approach, handling conflicts through atomic add.
 *
 * Non-symmetric matrix case.
 *
 * parameters:
 *   n_edges  <-- local number of internal graph edges (mesh faces)
 *   edges    <-- edges (mesh face -> cells) connectivity
 *   xa       <-- extradiagonal values
 *   x        <-- vector
 *   y        <-> vector
 *----------------------------------------------------------------------------*/

__global__ static void
_mat_vect_p_l_native_exdiag(cs_lnum_t           n_edges,
                            const cs_lnum_2_t  * __restrict__ edges,
                            const cs_real_t    *__restrict__ xa,
                            const cs_real_t    *__restrict__ x,
                            cs_real_t          *__restrict__ y)
{
  cs_lnum_t edge_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (edge_id < n_edges) {
    cs_lnum_t ii = edges[edge_id][0];
    cs_lnum_t jj = edges[edge_id][1];
    cs_real_t x_ii = __ldg(x + ii);
    cs_real_t x_jj = __ldg(x + jj);
    atomicAdd(&y[ii], xa[edge_id*2]     * x_jj);
    atomicAdd(&y[jj], xa[edge_id*2 + 1] * x_ii);
  }
}

/*----------------------------------------------------------------------------
 * SpMV extradiagonal terms using native to face-based array and scatter
 * approach, handling conflicts through atomic add.
 *
 * Symmetric matrix case.
 *
 * parameters:
 *   n_edges  <-- local number of internal graph edges (mesh faces)
 *   edges    <-- edges (mesh face -> cells) connectivity
 *   xa       <-- extradiagonal values
 *   x        <-- vector
 *   y        <-> vector
 *----------------------------------------------------------------------------*/

__global__ static void
_mat_vect_p_l_native_exdiag_sym(cs_lnum_t           n_edges,
                                const cs_lnum_2_t  * __restrict__ edges,
                                const cs_real_t    *__restrict__ xa,
                                const cs_real_t    *__restrict__ x,
                                cs_real_t          *__restrict__ y)
{
  cs_lnum_t edge_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (edge_id < n_edges) {
    cs_lnum_t ii = edges[edge_id][0];
    cs_lnum_t jj = edges[edge_id][1];
    cs_real_t x_ii = __ldg(x + ii);
    cs_real_t x_jj = __ldg(x + jj);
    atomicAdd(&y[ii], xa[edge_id] * x_jj);
    atomicAdd(&y[jj], xa[edge_id] * x_ii);
  }
}

/*----------------------------------------------------------------------------*/
/* \brief Local matrix.vector product y = A.x with CSR matrix arrays.
 *
 * \param[in]   n_rows     number of local rows
 * \param[in]   row_index  pointer to matrix rows index
 * \param[in]   col_id     pointer to matrix column id
 * \param[in]   val        pointer to matrix values
 * \param[in]   x          multipliying vector values
 * \param[out]  y          resulting vector
 */
/*----------------------------------------------------------------------------*/

__global__ static void
_mat_vect_p_l_csr(cs_lnum_t         n_rows,
                  const cs_lnum_t  *__restrict__ row_index,
                  const cs_lnum_t  *__restrict__ col_id,
                  const cs_real_t  *__restrict__ val,
                  const cs_real_t  *__restrict__ x,
                  cs_real_t        *__restrict__ y)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;
  cs_lnum_t jj;

  if (ii < n_rows) {
    cs_real_t sii = 0.0;
    const cs_lnum_t *__restrict__ _col_id = col_id + row_index[ii];
    const cs_real_t *__restrict__ m_row  = val + row_index[ii];
    cs_lnum_t n_cols = row_index[ii + 1] - row_index[ii];
#pragma unroll
    for (jj = 0; jj < n_cols; jj++) {
      sii += m_row[jj] * __ldg(x + _col_id[jj]);
    }
    y[ii] = sii;
  }
}

/*----------------------------------------------------------------------------*/
/* \brief Local matrix.vector product y = A.x with CSR matrix arrays,
 *        excluding diagonal part.
 *
 * \param[in]   n_rows     number of local rows
 * \param[in]   row_index  pointer to matrix rows index
 * \param[in]   col_id     pointer to matrix column id
 * \param[in]   val        pointer to matrix values
 * \param[in]   x          multipliying vector values
 * \param[out]  y          resulting vector
 */
/*----------------------------------------------------------------------------*/

__global__ static void
_mat_vect_p_l_csr_exdiag(cs_lnum_t         n_rows,
                         const cs_lnum_t  *__restrict__ row_index,
                         const cs_lnum_t  *__restrict__ col_id,
                         const cs_real_t  *__restrict__ val,
                         const cs_real_t  *__restrict__ x,
                         cs_real_t        *__restrict__ y)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;

  if (ii < n_rows) {
    cs_real_t        sii            = 0.0;
    const cs_lnum_t *__restrict__ _col_id = col_id + row_index[ii];
    const cs_real_t *__restrict__ m_row  = val + row_index[ii];
    cs_lnum_t n_cols = row_index[ii + 1] - row_index[ii];
#pragma unroll
    for (cs_lnum_t jj = 0; jj < n_cols; jj++) {
      cs_lnum_t c_id = _col_id[jj];
      if (c_id != ii)
        sii += m_row[jj] * __ldg(x + c_id);
    }
    y[ii] = sii;
  }
}

/*----------------------------------------------------------------------------*/
/* \brief Substract local diagonal contribution with CSR matrix arrays.
 *
 * \param[in]   n_rows     number of local rows
 * \param[in]   row_index  pointer to matrix rows index
 * \param[in]   col_id     pointer to matrix column id
 * \param[in]   val        pointer to matrix values
 * \param[in]   x          multipliying vector values
 * \param[out]  y          resulting vector
 */
/*----------------------------------------------------------------------------*/

__global__ static void
_mat_vect_p_l_csr_substract_diag(cs_lnum_t         n_rows,
                                 const cs_lnum_t  *__restrict__ row_index,
                                 const cs_lnum_t  *__restrict__ col_id,
                                 const cs_real_t  *__restrict__ val,
                                 const cs_real_t  *__restrict__ x,
                                 cs_real_t        *__restrict__ y)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;

  if (ii < n_rows) {
    const cs_lnum_t *__restrict__ _col_id = col_id + row_index[ii];
    const cs_real_t *__restrict__ m_row  = val + row_index[ii];
    cs_lnum_t n_cols = row_index[ii + 1] - row_index[ii];
#pragma unroll
    for (cs_lnum_t jj = 0; jj < n_cols; jj++) {
      cs_lnum_t c_id = _col_id[jj];
      if (c_id == ii) {
        y[ii] -= m_row[jj] * x[ii];
        break;
      }
    }
  }
}

/*----------------------------------------------------------------------------*/
/* \brief Local matrix.vector product y = A.x with MSR matrix arrays.
 *
 * \param[in]   n_rows     number of local rows
 * \param[in]   row_index  pointer to matrix rows index
 * \param[in]   col_id     pointer to matrix column id
 * \param[in]   d_val      pointer to diagonal matrix values
 * \param[in]   x_val      pointer to extradiagonal matrix values
 * \param[in]   x          multipliying vector values
 * \param[out]  y          resulting vector
 */
/*----------------------------------------------------------------------------*/

__global__ static void
_mat_vect_p_l_msr(cs_lnum_t         n_rows,
                  const cs_lnum_t  *__restrict__ row_index,
                  const cs_lnum_t  *__restrict__ col_id,
                  const cs_real_t  *__restrict__ d_val,
                  const cs_real_t  *__restrict__ x_val,
                  const cs_real_t  *__restrict__ x,
                  cs_real_t        *__restrict__ y)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;

  if (ii < n_rows) {
    const cs_lnum_t *__restrict__ _col_id = col_id + row_index[ii];
    const cs_real_t *__restrict__ m_row  = x_val + row_index[ii];

    cs_lnum_t n_cols = row_index[ii + 1] - row_index[ii];

    cs_real_t sii = 0.0;

    for (cs_lnum_t jj = 0; jj < n_cols; jj++)
      sii += m_row[jj] * __ldg(x + _col_id[jj]);

    y[ii] = sii + d_val[ii] * x[ii];
  }
}

/*----------------------------------------------------------------------------*/
/* \brief Local diagonal contribution y = Da.x  + y.
 *
 * This can be combined with a cuSPARSE CSR SpMV product with the
 * extra-diagonal portion of an MSR or distributed matrix.
 *
 * \param[in]       n_rows  number of local rows
 * \param[in]       d_val   pointer to diagonal matrix values
 * \param[in]       x       multipliying vector values
 * \param[in, out]  y       resulting vector
 */
/*----------------------------------------------------------------------------*/

__global__ static void
_mat_vect_p_l_msr_diag(cs_lnum_t         n_rows,
                       const cs_real_t  *__restrict__ d_val,
                       const cs_real_t  *__restrict__ x,
                       cs_real_t        *__restrict__ y)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;

  if (ii < n_rows)
    y[ii] = d_val[ii] * x[ii];
}

/*----------------------------------------------------------------------------*/
/* \brief Local matrix.vector product y = A.x with MSR matrix,
 *        3x3 blocked diagonal version.
 *
 * \param[in]   n_rows     number of local rows
 * \param[in]   d_val      pointer to diagonal matrix values
 * \param[in]   x          multipliying vector values
 * \param[out]  y          resulting vector
 */
/*----------------------------------------------------------------------------*/

__global__ static void
_b_3_3_spmv_diag(cs_lnum_t        n_rows,
                 const cs_real_t  *__restrict__ d_val,
                 const cs_real_t  *__restrict__ x,
                 cs_real_t        *__restrict__ y)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;
  if (ii < n_rows) {

#   pragma unroll
    for (cs_lnum_t kk = 0; kk < 3; kk++) {
      y[ii*3 + kk] =   d_val[ii * 9 + kk * 3]     * x[ii * 3]
                     + d_val[ii * 9 + kk * 3 + 1] * x[ii * 3 + 1]
                     + d_val[ii * 9 + kk * 3 + 2] * x[ii * 3 + 2];
    }

  }
}

/*----------------------------------------------------------------------------*/
/* \brief Local matrix.vector product y = A.x with MSR matrix,
 *        templated blocked diagonal version.
 *
 * \param[in]   n_rows     number of local rows
 * \param[in]   d_val      pointer to diagonal matrix values
 * \param[in]   x          multipliying vector values
 * \param[out]  y          resulting vector
 */
/*----------------------------------------------------------------------------*/

template <const int n>
__global__ static void
_b_spmv_diag(cs_lnum_t        n_rows,
             const cs_real_t  *__restrict__ d_val,
             const cs_real_t  *__restrict__ x,
             cs_real_t        *__restrict__ y)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;

  if (ii < n_rows) {
    const cs_lnum_t nn = n*n;

    cs_real_t sii[n];

    for (cs_lnum_t kk = 0; kk < n; kk++)
      sii[kk] = 0.;

    for (cs_lnum_t kk = 0; kk < n; kk++) {
      for (cs_lnum_t ll = 0; ll < n; ll++) {
        sii[kk] += d_val[ii*nn + kk*n + ll] * x[ii*n + ll];
      }
    }

    for (cs_lnum_t kk = 0; kk < n; kk++)
      y[ii*n + kk] = sii[kk];
  }
}

/*----------------------------------------------------------------------------*/
/* \brief Local matrix.vector product y = A.x with MSR matrix,
 *        3x3 blocked diagonal version.
 *
 * \param[in]   n_rows     number of local rows
 * \param[in]   row_index  pointer to matrix rows index
 * \param[in]   col_id     pointer to matrix column id
 * \param[in]   d_val      pointer to diagonal matrix values
 * \param[in]   x_val      pointer to extradiagonal matrix values
 * \param[in]   x          multipliying vector values
 * \param[out]  y          resulting vector
 */
/*----------------------------------------------------------------------------*/

__global__ static void
_b_3_3_mat_vect_p_l_msr(cs_lnum_t        n_rows,
                        const cs_lnum_t  *__restrict__ col_id,
                        const cs_lnum_t  *__restrict__ row_index,
                        const cs_real_t  *__restrict__ d_val,
                        const cs_real_t  *__restrict__ x_val,
                        const cs_real_t  *__restrict__ x,
                        cs_real_t        *__restrict__ y)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;

  if (ii < n_rows) {
    const cs_lnum_t *__restrict__ _col_id = col_id + row_index[ii];
    const cs_real_t *__restrict__ m_row  = x_val + row_index[ii];
    cs_lnum_t n_cols = row_index[ii + 1] - row_index[ii];
    cs_real_t sii[3];
    for (cs_lnum_t kk = 0; kk < 3; kk++) {
      sii[kk] =   d_val[ii * 9 + kk * 3]     * x[ii * 3]
                + d_val[ii * 9 + kk * 3 + 1] * x[ii * 3 + 1]
                + d_val[ii * 9 + kk * 3 + 2] * x[ii * 3 + 2];
    }

    for (cs_lnum_t jj = 0; jj < n_cols; jj++) {
      for (cs_lnum_t kk = 0; kk < 3; kk++)
        sii[kk] += m_row[jj] * __ldg(x + (_col_id[jj]*3 + kk));
    }

    y[ii*3]     = sii[0];
    y[ii*3 + 1] = sii[1];
    y[ii*3 + 2] = sii[2];
  }
}

/*----------------------------------------------------------------------------*/
/* \brief Local matrix.vector product y = A.x with MSR matrix,
 *        excluding 3x3 blocked diagonal.
 *
 * \param[in]   n_rows     number of local rows
 * \param[in]   row_index  pointer to matrix rows index
 * \param[in]   col_id     pointer to matrix column id
 * \param[in]   d_val      pointer to diagonal matrix values
 * \param[in]   x_val      pointer to extradiagonal matrix values
 * \param[in]   x          multipliying vector values
 * \param[out]  y          resulting vector
 */
/*----------------------------------------------------------------------------*/

__global__ static void
_b_3_3_mat_vect_p_l_msr_exdiag(cs_lnum_t        n_rows,
                               const cs_lnum_t  *__restrict__ col_id,
                               const cs_lnum_t  *__restrict__ row_index,
                               const cs_real_t  *__restrict__ d_val,
                               const cs_real_t  *__restrict__ x_val,
                               const cs_real_t  *__restrict__ x,
                               cs_real_t        *__restrict__ y)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;

  if (ii < n_rows) {
    const cs_lnum_t *__restrict__ _col_id = col_id + row_index[ii];
    const cs_real_t *__restrict__ m_row  = x_val + row_index[ii];
    cs_lnum_t n_cols = row_index[ii + 1] - row_index[ii];
    cs_real_t sii[3];
    for (cs_lnum_t kk = 0; kk < 3; kk++)
      sii[kk] = 0.;

    for (cs_lnum_t jj = 0; jj < n_cols; jj++) {
      for (cs_lnum_t kk = 0; kk < 3; kk++)
        sii[kk] += m_row[jj] * __ldg(x + (_col_id[jj]*3 + kk));
    }

    y[ii * 3]     = sii[0];
    y[ii * 3 + 1] = sii[1];
    y[ii * 3 + 2] = sii[2];
  }
}

/*----------------------------------------------------------------------------*/
/* \brief Local matrix.vector product y = A.x with MSR matrix,
 *        blocked diagonal version.
 *
 * \param[in]   n_rows     number of local rows
 * \param[in]   row_index  pointer to matrix rows index
 * \param[in]   col_id     pointer to matrix column id
 * \param[in]   d_val      pointer to diagonal matrix values
 * \param[in]   x_val      pointer to extradiagonal matrix values
 * \param[in]   x          multipliying vector values
 * \param[out]  y          resulting vector
 */
/*----------------------------------------------------------------------------*/

template <const int n>
__global__ static void
_b_mat_vect_p_l_msr(cs_lnum_t        n_rows,
                    const cs_lnum_t  *__restrict__ col_id,
                    const cs_lnum_t  *__restrict__ row_index,
                    const cs_real_t  *__restrict__ d_val,
                    const cs_real_t  *__restrict__ x_val,
                    const cs_real_t  *__restrict__ x,
                    cs_real_t        *__restrict__ y)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;

  if (ii < n_rows) {
    const cs_lnum_t nn = n*n;

    const cs_lnum_t *__restrict__ _col_id = col_id + row_index[ii];
    const cs_real_t *__restrict__ m_row  = x_val + row_index[ii];
    cs_lnum_t n_cols = row_index[ii + 1] - row_index[ii];
    cs_real_t sii[n];

    for (cs_lnum_t kk = 0; kk < n; kk++)
      sii[kk] = 0.;

    for (cs_lnum_t kk = 0; kk < n; kk++) {
      for (cs_lnum_t ll = 0; ll < n; ll++) {
        sii[kk] += d_val[ii*nn + kk*n + ll] * x[ii*n + ll];
      }
    }

    for (cs_lnum_t jj = 0; jj < n_cols; jj++) {
      for (cs_lnum_t kk = 0; kk < n; kk++)
        sii[kk] += m_row[jj] * __ldg(x + (_col_id[jj]*n + kk));
    }

    for (cs_lnum_t kk = 0; kk < n; kk++)
      y[ii*n + kk] = sii[kk];
  }
}

/*----------------------------------------------------------------------------*/
/* \brief Local matrix.vector product y = A.x with MSR matrix,
 *        excluding blocked diagonal.
 *
 * \param[in]   n_rows     number of local rows
 * \param[in]   row_index  pointer to matrix rows index
 * \param[in]   col_id     pointer to matrix column id
 * \param[in]   d_val      pointer to diagonal matrix values
 * \param[in]   x_val      pointer to extradiagonal matrix values
 * \param[in]   x          multipliying vector values
 * \param[out]  y          resulting vector
 */
/*----------------------------------------------------------------------------*/

template <const int n>
__global__ static void
_b_mat_vect_p_l_msr_exdiag(cs_lnum_t        n_rows,
                           const cs_lnum_t  *__restrict__ col_id,
                           const cs_lnum_t  *__restrict__ row_index,
                           const cs_real_t  *__restrict__ d_val,
                           const cs_real_t  *__restrict__ x_val,
                           const cs_real_t  *__restrict__ x,
                           cs_real_t        *__restrict__ y)
{
  cs_lnum_t ii = blockIdx.x * blockDim.x + threadIdx.x;

  if (ii < n_rows) {
    const cs_lnum_t *__restrict__ _col_id = col_id + row_index[ii];
    const cs_real_t *__restrict__ m_row  = x_val + row_index[ii];
    cs_lnum_t n_cols = row_index[ii + 1] - row_index[ii];
    cs_real_t sii[n];
    for (cs_lnum_t kk = 0; kk < n; kk++)
      sii[kk] = 0.;

    for (cs_lnum_t jj = 0; jj < n_cols; jj++) {
      for (cs_lnum_t kk = 0; kk < n; kk++)
        sii[kk] += m_row[jj] * __ldg(x + (_col_id[jj]*n + kk));
    }

    for (cs_lnum_t kk = 0; kk < n; kk++)
      y[ii*n + kk] = sii[kk];
  }
}

/*----------------------------------------------------------------------------
 * Start synchronization of ghost values prior to matrix.vector product.
 *
 * Values are packed on the device, so:
 * - If MPI is CUDA-aware, no values need to go through the host
 * - Otherwise, only halo values need to go through the host, not the
 *   whole array.
 *
 * parameters:
 *   matrix   <-- pointer to matrix structure
 *   d_x      <-> multipliying vector values (ghost values updated)
 *
 * returns:
 *   halo state to use for synchronisation finalisation.
 *----------------------------------------------------------------------------*/

static cs_halo_state_t *
_pre_vector_multiply_sync_x_start(const cs_matrix_t   *matrix,
                                  cs_real_t            d_x[restrict])
{
  cs_halo_state_t *hs = NULL;

  if (matrix->halo != NULL) {

    if (_stream != 0)
      hipStreamSynchronize(_stream);

    hs = cs_halo_state_get_default();

    cs_halo_sync_pack_d(matrix->halo,
                        CS_HALO_STANDARD,
                        CS_REAL_TYPE,
                        matrix->db_size,
                        d_x,
                        NULL,
                        hs);

    cs_halo_sync_start(matrix->halo, d_x, hs);

  }

  return hs;
}

#if defined(HAVE_CUSPARSE)

/*----------------------------------------------------------------------------
 * Unset matrix cuSPARSE mapping.
 *
 * parameters:
 *   matrix    <-- pointer to matrix structure
 *----------------------------------------------------------------------------*/

static void
_unset_cusparse_map(cs_matrix_t   *matrix)
{
  cs_matrix_cusparse_map_t *csm
    = (cs_matrix_cusparse_map_t *)matrix->ext_lib_map;

  if (csm == NULL)
    return;

#if defined(HAVE_CUSPARSE_GENERIC_API)

  hipsparseDestroySpMat(csm->matA);

  if (csm->block_diag == false) {
    if (csm->vecXValues != NULL)
      hipsparseDestroyDnVec(csm->vecX);
    if (csm->vecYValues != NULL)
      hipsparseDestroyDnVec(csm->vecY);
  }
  else {
    if (csm->vecXValues != NULL)
      hipsparseDestroyDnMat(csm->matX);
    if (csm->vecYValues != NULL)
      hipsparseDestroyDnMat(csm->matY);
  }

  if (csm->dBuffer != NULL) {
    CS_CUDA_CHECK(hipFree(csm->dBuffer));
    csm->dBuffer = NULL;
  }

  csm->block_diag = false;

  csm->vecXValues = NULL;
  csm->vecYValues = NULL;

#else

  hipsparseDestroyMatDescr(csm->descrA);

#endif

  csm->nnz = 0;
  csm->d_row_index = NULL;
  csm->d_col_id = NULL;
  csm->d_e_val = NULL;

  BFT_FREE(matrix->ext_lib_map);
  matrix->destroy_adaptor = NULL;
}

/*----------------------------------------------------------------------------
 * Set matrix cuSPARSE mapping.
 *
 * parameters:
 *   matrix    <-- pointer to matrix structure
 *----------------------------------------------------------------------------*/

static cs_matrix_cusparse_map_t *
_set_cusparse_map(cs_matrix_t   *matrix)
{
  cs_matrix_cusparse_map_t *csm
    = (cs_matrix_cusparse_map_t *)matrix->ext_lib_map;

  if (csm != NULL) {
    _unset_cusparse_map(matrix);
  }
  else {
    BFT_MALLOC(csm, 1, cs_matrix_cusparse_map_t);
    matrix->ext_lib_map = (void *)csm;
  }
  matrix->destroy_adaptor = _unset_cusparse_map;

  const void *row_index, *col_id;
  const void *e_val;
  cs_lnum_t nnz = 0;

  if (matrix->type == CS_MATRIX_CSR) {
    const cs_matrix_struct_csr_t *ms
      = (const cs_matrix_struct_csr_t  *)matrix->structure;
    const cs_matrix_coeff_csr_t *mc
      = (const cs_matrix_coeff_csr_t *)matrix->coeffs;
    nnz = ms->row_index[matrix->n_rows];
    row_index = cs_get_device_ptr_const_pf
                  (const_cast<cs_lnum_t *>(ms->row_index));
    col_id = cs_get_device_ptr_const_pf
               (const_cast<cs_lnum_t *>(ms->col_id));
    e_val = cs_get_device_ptr_const_pf
              (const_cast<cs_real_t *>(mc->val));
  }
  else {
    const cs_matrix_struct_dist_t *ms
      = (const cs_matrix_struct_dist_t *)matrix->structure;
    const cs_matrix_coeff_dist_t *mc
      = (const cs_matrix_coeff_dist_t *)matrix->coeffs;
    nnz = ms->e.row_index[matrix->n_rows];
    row_index = cs_get_device_ptr_const_pf
                  (const_cast<cs_lnum_t *>(ms->e.row_index));
    col_id = cs_get_device_ptr_const_pf
               (const_cast<cs_lnum_t *>(ms->e.col_id));
    e_val = cs_get_device_ptr_const_pf
              (const_cast<cs_real_t *>(mc->e_val));
  }

  hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;

  if (_handle == NULL)
    status = hipsparseCreate(&_handle);

#if defined(HAVE_CUSPARSE_GENERIC_API)

  if (matrix->db_size > matrix->eb_size)
    csm->block_diag = true;
  else
    csm->block_diag = false;

  if (HIPSPARSE_STATUS_SUCCESS != status)
    bft_error(__FILE__, __LINE__, 0, _("%s: %s."),
              __func__, hipsparseGetErrorString(status));

  csm->vecXValues = NULL;  /* Pointer to vector values */
  csm->vecYValues = NULL;  /* Pointer to vector values */
  csm->dBuffer = NULL;

  csm->d_e_val = NULL;
  csm->d_row_index = NULL;
  csm->d_col_id = NULL;

  if (matrix->eb_size == 1) {

    hipsparseIndexType_t index_dtype
      = (sizeof(cs_lnum_t) == 4) ? HIPSPARSE_INDEX_32I : HIPSPARSE_INDEX_64I;
    hipDataType val_dtype
      = (sizeof(cs_real_t) == 8) ? HIP_R_64F : HIP_R_32F;

    status = hipsparseCreateCsr(&(csm->matA),
                               matrix->n_rows,
                               matrix->n_cols_ext,
                               nnz,
                               const_cast<void *>(row_index),
                               const_cast<void *>(col_id),
                               const_cast<void *>(e_val),
                               index_dtype,
                               index_dtype,
                               HIPSPARSE_INDEX_BASE_ZERO,
                               val_dtype);

  }
  else {

    csm->nnz = nnz;
    csm->d_e_val = const_cast<void *>(e_val);

    csm->d_row_index = const_cast<void *>(row_index);
    csm->d_col_id = const_cast<void *>(col_id);

    status = hipsparseCreateMatDescr(&(csm->descrA));

  }

  if (HIPSPARSE_STATUS_SUCCESS != status)
    bft_error(__FILE__, __LINE__, 0, _("%s: %s."),
              __func__, hipsparseGetErrorString(status));

#else

  if (HIPSPARSE_STATUS_SUCCESS != status)
    bft_error(__FILE__, __LINE__, 0, _("%s: cuSPARSE error %d."),
              __func__, (int)status);

  csm->nnz = nnz;
  csm->d_e_val = const_cast<void *>(e_val);

  csm->d_row_index = const_cast<void *>(row_index);
  csm->d_col_id = const_cast<void *>(col_id);

  status = hipsparseCreateMatDescr(&(csm->descrA));

  if (HIPSPARSE_STATUS_SUCCESS != status)
    bft_error(__FILE__, __LINE__, 0, _("%s: cuSPARSE error %d."),
              __func__, (int)status);

  hipsparseSetMatIndexBase(csm->descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(csm->descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatDiagType(csm->descrA, HIPSPARSE_DIAG_TYPE_NON_UNIT);

#endif

  return csm;
}

/*----------------------------------------------------------------------------
 * Update matrix cuSPARSE mapping.
 *
 * parameters:
 *   csm       <-> cuSPARSE matrix mapping
 *   matrix    <-- pointer to matrix structure
 *   d_x       <-- pointer to input vector (on device)
 *   d_y       <-- pointer to output vector (on device)
 *----------------------------------------------------------------------------*/

static void
_update_cusparse_map(cs_matrix_cusparse_map_t  *csm,
                     const cs_matrix_t         *matrix,
                     void                      *d_x,
                     void                      *d_y)
{
  assert(csm != NULL);

#if defined(HAVE_CUSPARSE_GENERIC_API)

#if CUSPARSE_VER_MAJOR >= 12
  hipsparseSpMVAlg_t spmv_alg_type = HIPSPARSE_SPMV_ALG_DEFAULT;
#else
  hipsparseSpMVAlg_t spmv_alg_type = HIPSPARSE_MV_ALG_DEFAULT;
#endif

  hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;
  hipDataType val_dtype
    = (sizeof(cs_real_t) == 8) ? HIP_R_64F : HIP_R_32F;

  if (d_x != csm->vecXValues) {
    if (csm->vecXValues != NULL)
      hipsparseDestroyDnVec(csm->vecX);

    status = hipsparseCreateDnVec(&(csm->vecX),
                                 matrix->n_cols_ext,
                                 d_x,
                                 val_dtype);

    if (HIPSPARSE_STATUS_SUCCESS != status)
      bft_error(__FILE__, __LINE__, 0, _("%s: %s."),
                __func__, hipsparseGetErrorString(status));

    csm->vecXValues = d_x;
  }

  if (d_y != csm->vecYValues) {
    if (csm->vecYValues != NULL)
      hipsparseDestroyDnVec(csm->vecY);

    status = hipsparseCreateDnVec(&(csm->vecY),
                                 matrix->n_rows,
                                 d_y,
                                 val_dtype);

    if (HIPSPARSE_STATUS_SUCCESS != status)
      bft_error(__FILE__, __LINE__, 0, _("%s: %s."),
                __func__, hipsparseGetErrorString(status));

    csm->vecYValues = d_y;
  }

  if (csm->dBuffer == NULL) {
    size_t bufferSize = 0;
    cs_real_t alpha = 1.0;
    cs_real_t beta = 1.0;  /* 0 should be enough for SmPV, 1 needed for
                              y = A.x + b.y
                              which is useful when y is initialized by
                              a separate diagonal da.x product */

    status = hipsparseSpMV_bufferSize(_handle,
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     &alpha,
                                     csm->matA,
                                     csm->vecX,
                                     &beta,
                                     csm->vecY,
                                     val_dtype,
                                     spmv_alg_type,
                                     &bufferSize);

    CS_CUDA_CHECK(hipMalloc(&(csm->dBuffer), bufferSize));
  }

#endif
}

#if defined(HAVE_CUSPARSE_GENERIC_API)

/*----------------------------------------------------------------------------
 * Update matrix cuSPARSE mapping in block diagonal case.
 *
 * parameters:
 *   csm       <-> cuSPARSE matrix mapping
 *   matrix    <-- pointer to matrix structure
 *   d_x       <-- pointer to input vector (on device)
 *   d_y       <-- pointer to output vector (on device)
 *----------------------------------------------------------------------------*/

static void
_update_cusparse_map_block_diag(cs_matrix_cusparse_map_t  *csm,
                                const cs_matrix_t         *matrix,
                                void                      *d_x,
                                void                      *d_y)
{
  assert(csm != NULL);

  hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;
  hipDataType val_dtype
    = (sizeof(cs_real_t) == 8) ? HIP_R_64F : HIP_R_32F;

  if (d_x != csm->vecXValues) {
    if (csm->vecXValues != NULL)
      hipsparseDestroyDnMat(csm->matX);

    status = hipsparseCreateDnMat(&(csm->matX),
                                 matrix->n_cols_ext,
                                 matrix->db_size,
                                 matrix->db_size,
                                 d_x,
                                 val_dtype,
                                 HIPSPARSE_ORDER_ROW);

    if (HIPSPARSE_STATUS_SUCCESS != status)
      bft_error(__FILE__, __LINE__, 0, _("%s: %s."),
                __func__, hipsparseGetErrorString(status));

    csm->vecXValues = d_x;
  }

  if (d_y != csm->vecYValues) {
    if (csm->vecYValues != NULL)
      hipsparseDestroyDnMat(csm->matY);

    status = hipsparseCreateDnMat(&(csm->matY),
                                 matrix->n_rows,
                                 matrix->db_size,
                                 matrix->db_size,
                                 d_y,
                                 val_dtype,
                                 HIPSPARSE_ORDER_ROW);

    if (HIPSPARSE_STATUS_SUCCESS != status)
      bft_error(__FILE__, __LINE__, 0, _("%s: %s."),
                __func__, hipsparseGetErrorString(status));

    csm->vecYValues = d_y;
  }

  if (csm->dBuffer == NULL) {
    size_t bufferSize = 0;
    cs_real_t alpha = 1.0;
    cs_real_t beta = 1.0;  /* 0 should be enough for SmPV, 1 needed for
                              y = A.x + b.y
                              which is useful when y is initialized by
                              a separate diagonal da.x product */

    status = hipsparseSpMM_bufferSize(_handle,
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     &alpha,
                                     csm->matA,
                                     csm->matX,
                                     &beta,
                                     csm->matY,
                                     val_dtype,
                                     HIPSPARSE_SPMM_ALG_DEFAULT,
                                     &bufferSize);

    if (HIPSPARSE_STATUS_SUCCESS != status)
      bft_error(__FILE__, __LINE__, 0, _("%s: %s."),
                __func__, hipsparseGetErrorString(status));

    CS_CUDA_CHECK(hipMalloc(&(csm->dBuffer), bufferSize));
  }

}

#endif // defined(HAVE_CUSPARSE_GENERIC_API)

#endif // defined(HAVE_CUSPARSE)

/*! (DOXYGEN_SHOULD_SKIP_THIS) \endcond */

BEGIN_C_DECLS

/*============================================================================
 * Public function definitions
 *============================================================================*/

/*----------------------------------------------------------------------------*/
/*!
 * \brief Finalize CUDA matrix API.
 *
 * This frees resources such as the cuSPARSE handle, if used.
 */
/*----------------------------------------------------------------------------*/

void
cs_matrix_spmv_cuda_finalize(void)
{
  _stream = 0;

#if defined(HAVE_CUSPARSE)

  if (_handle != NULL) {
    hipsparseDestroy(_handle);
    _handle = NULL;
  }

#endif
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Assign CUDA stream for next CUDA-based SpMV operations.
 *
 * If a stream other than the default stream (0) is used, it will not be
 * synchronized automatically after sparse matrix-vector products (so as to
 * avoid the corresponding overhead), so the caller will need to manage
 * stream syncronization manually.
 *
 * This function is callable only from CUDA code.
 */
/*----------------------------------------------------------------------------*/

void
cs_matrix_spmv_cuda_set_stream(hipStream_t  stream)
{
  _stream = stream;
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Return stream used for CUDA-based SpMV operations.
 *
 * This function is callable only from CUDA code.
 */
/*----------------------------------------------------------------------------*/

hipStream_t
cs_matrix_spmv_cuda_get_stream(void)
{
  return _stream;
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Matrix.vector product y = A.x with MSR matrix, scalar CUDA version.
 *
 * \param[in]   matrix        pointer to matrix structure
 * \param[in]   exclude_diag  exclude diagonal if true,
 * \param[in]   sync          synchronize ghost cells if true
 * \param[in]   d_x           multipliying vector values (on device)
 * \param[out]  d_y           resulting vector (on device)
 */
/*----------------------------------------------------------------------------*/

void
cs_matrix_spmv_cuda_native(const cs_matrix_t  *matrix,
                           bool                exclude_diag,
                           bool                sync,
                           cs_real_t           d_x[restrict],
                           cs_real_t           d_y[restrict])
{
  const cs_matrix_struct_native_t  *ms
    = (const cs_matrix_struct_native_t *)matrix->structure;

  const cs_matrix_coeff_dist_t  *mc
    = (const cs_matrix_coeff_dist_t *)matrix->coeffs;

  const cs_real_t *__restrict__ da
    = (const cs_real_t *)cs_get_device_ptr_const_pf
                           (const_cast<cs_real_t *>(mc->d_val));
  const cs_real_t *__restrict__ xa
    = (const cs_real_t *)cs_get_device_ptr_const_pf
                           (const_cast<cs_real_t *>(mc->e_val));

  /* Ghost cell communication */

  cs_halo_state_t *hs = NULL;
  if (sync)
    hs = _pre_vector_multiply_sync_x_start(matrix, d_x);

  /* Diagonal part of matrix.vector product */

  unsigned int blocksize = 256;
  unsigned int gridsize = cs_cuda_grid_size(ms->n_cols_ext, blocksize);

  if (!exclude_diag)
    _mat_vect_p_l_native_diag<<<gridsize, blocksize, 0, _stream>>>
      (ms->n_rows, ms->n_cols_ext, da, d_x, d_y);
  else
    _zero_range<<<gridsize, blocksize, 0, _stream>>>
      (ms->n_cols_ext, d_y);

  /* Finalize ghost cell comunication if overlap used */

  if (hs != NULL)
    cs_halo_sync_wait(matrix->halo, d_x, hs);

  hipStreamSynchronize(_stream);

  /* Non-diagonal terms */

  if (xa != NULL) {
    gridsize = cs_cuda_grid_size(ms->n_edges, blocksize);

    const cs_lnum_2_t *restrict edges
      = (const cs_lnum_2_t *)cs_get_device_ptr_const_pf
                               (const_cast<cs_lnum_2_t *>(ms->edges));

#if 1
    if (mc->symmetric)
      _mat_vect_p_l_native_exdiag_sym<<<gridsize, blocksize, 0, _stream>>>
        (ms->n_edges, edges, xa, d_x, d_y);
    else
      _mat_vect_p_l_native_exdiag<<<gridsize, blocksize, 0, _stream>>>
        (ms->n_edges, edges, xa, d_x, d_y);

#else
    if (mc->symmetric) {
      for (cs_lnum_t e_id = 0; e_id < ms->n_edges; e_id++) {
        cs_lnum_t ii = edges[e_id][0];
        cs_lnum_t jj = edges[e_id][1];
        d_y[ii] += xa[e_id] * d_x[jj];
        d_y[jj] += xa[e_id] * d_x[ii];
      }

    }
    else {
      for (cs_lnum_t e_id = 0; e_id < ms->n_edges; e_id++) {
        cs_lnum_t ii = edges[e_id][0];
        cs_lnum_t jj = edges[e_id][1];
        d_y[ii] += xa[2*e_id] * d_x[jj];
        d_y[jj] += xa[2*e_id + 1] * d_x[ii];
      }
    }
#endif
  }

  hipStreamSynchronize(_stream);

  if (_stream == 0) {
    hipStreamSynchronize(0);
    CS_CUDA_CHECK(hipGetLastError());
  }
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Matrix.vector product y = A.x with CSR matrix, scalar CUDA version.
 *
 * \param[in]   matrix        pointer to matrix structure
 * \param[in]   exclude_diag  exclude diagonal if true,
 * \param[in]   sync          synchronize ghost cells if true
 * \param[in]   d_x           multipliying vector values (on device)
 * \param[out]  d_y           resulting vector (on device)
 */
/*----------------------------------------------------------------------------*/

void
cs_matrix_spmv_cuda_csr(cs_matrix_t  *matrix,
                        bool          exclude_diag,
                        bool          sync,
                        cs_real_t     d_x[restrict],
                        cs_real_t     d_y[restrict])
{
  const cs_matrix_struct_csr_t *ms
    = (const cs_matrix_struct_csr_t *)matrix->structure;
  const cs_matrix_coeff_csr_t *mc
    = (const cs_matrix_coeff_csr_t  *)matrix->coeffs;

  const cs_lnum_t *__restrict__ row_index
    = (const cs_lnum_t *)cs_get_device_ptr_const_pf
                           (const_cast<cs_lnum_t *>(ms->row_index));
  const cs_lnum_t *__restrict__ col_id
    = (const cs_lnum_t *)cs_get_device_ptr_const_pf
                           (const_cast<cs_lnum_t *>(ms->col_id));
  const cs_real_t *__restrict__ val
    = (const cs_real_t *)cs_get_device_ptr_const_pf
                           (const_cast<cs_real_t *>(mc->val));

  /* Ghost cell communication */

  if (sync) {
    cs_halo_state_t *hs = _pre_vector_multiply_sync_x_start(matrix, d_x);
    cs_halo_sync_wait(matrix->halo, d_x, hs);
  }

  /* Compute SpMV */

  unsigned int blocksize = 256;
  unsigned int gridsize
    = (unsigned int)ceil((double)ms->n_rows / blocksize);

  if (!exclude_diag)
    _mat_vect_p_l_csr<<<gridsize, blocksize, 0, _stream>>>
      (ms->n_rows, row_index, col_id, val, d_x, d_y);
  else
    _mat_vect_p_l_csr_exdiag<<<gridsize, blocksize, 0, _stream>>>
      (ms->n_rows, row_index, col_id, val, d_x, d_y);

  if (_stream == 0) {
    hipStreamSynchronize(0);
    CS_CUDA_CHECK(hipGetLastError());
  }
}

#if defined(HAVE_CUSPARSE)

/*----------------------------------------------------------------------------*/
/*!
 * \brief Matrix.vector product y = A.x with CSR matrix, scalar cuSPARSE version.
 *
 * \param[in]   matrix        pointer to matrix structure
 * \param[in]   exclude_diag  exclude diagonal if true,
 * \param[in]   sync          synchronize ghost cells if true
 * \param[in]   d_x           multipliying vector values (on device)
 * \param[out]  d_y           resulting vector (on device)
 */
/*----------------------------------------------------------------------------*/

void
cs_matrix_spmv_cuda_csr_cusparse(cs_matrix_t  *matrix,
                                 bool          exclude_diag,
                                 bool          sync,
                                 cs_real_t     d_x[restrict],
                                 cs_real_t     d_y[restrict])
{
  cs_matrix_cusparse_map_t *csm
    = (cs_matrix_cusparse_map_t *)matrix->ext_lib_map;

  if (csm == NULL) {
    matrix->ext_lib_map = _set_cusparse_map(matrix);
    csm = (cs_matrix_cusparse_map_t *)matrix->ext_lib_map;
  }

  /* Ghost cell communication */

  if (sync) {
    cs_halo_state_t *hs = _pre_vector_multiply_sync_x_start(matrix,
                                                            (cs_real_t *)d_x);
    cs_halo_sync_wait(matrix->halo, (cs_real_t *)d_x, hs);
  }

  _update_cusparse_map(csm, matrix, d_x, d_y);

  cs_real_t alpha = 1.0;
  cs_real_t beta = 0.0;

  hipsparseSetStream(_handle, _stream);

  hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;

#if defined(HAVE_CUSPARSE_GENERIC_API)

#if CUSPARSE_VER_MAJOR >= 12
  hipsparseSpMVAlg_t spmv_alg_type = HIPSPARSE_SPMV_ALG_DEFAULT;
#else
  hipsparseSpMVAlg_t spmv_alg_type = HIPSPARSE_MV_ALG_DEFAULT;
#endif

  hipDataType val_dtype
    = (sizeof(cs_real_t) == 8) ? HIP_R_64F : HIP_R_32F;

  status = hipsparseSpMV(_handle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &alpha,
                        csm->matA,
                        csm->vecX,
                        &beta,
                        csm->vecY,
                        val_dtype,
                        spmv_alg_type,
                        csm->dBuffer);

  if (HIPSPARSE_STATUS_SUCCESS != status)
    bft_error(__FILE__, __LINE__, 0, _("%s: %s."),
              __func__, hipsparseGetErrorString(status));

#else

  if (sizeof(cs_real_t) == 8) {
    double _alpha = alpha;
    double _beta = beta;

    status = hipsparseDcsrmv(_handle,
                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            matrix->n_rows,
                            matrix->n_cols_ext,
                            csm->nnz,
                            &_alpha,
                            csm->descrA,
                            (const double *)csm->d_e_val,
                            (const int *)csm->d_row_index,
                            (const int *)csm->d_col_id,
                            (const double *)d_x,
                            &_beta,
                            (double *)d_y);
  }

  else if (sizeof(cs_real_t) == 4) {
    float _alpha = alpha;
    float _beta = beta;

    status = hipsparseScsrmv(_handle,
                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            matrix->n_rows,
                            matrix->n_cols_ext,
                            csm->nnz,
                            &_alpha,
                            csm->descrA,
                            (const float *)csm->d_e_val,
                            (const int *)csm->d_row_index,
                            (const int *)csm->d_col_id,
                            (const float *)d_x,
                            &_beta,
                            (float *)d_y);

  if (HIPSPARSE_STATUS_SUCCESS != status)
    bft_error(__FILE__, __LINE__, 0, _("%s: cuSPARSE error %d."),
              __func__, (int)status);

  }

#endif

  if (exclude_diag) {

    const cs_matrix_struct_csr_t *ms
      = (const cs_matrix_struct_csr_t *)matrix->structure;
    const cs_matrix_coeff_csr_t *mc
      = (const cs_matrix_coeff_csr_t  *)matrix->coeffs;
    const cs_lnum_t *__restrict__ d_row_index
      = (const cs_lnum_t *)cs_get_device_ptr_const_pf
                             (const_cast<cs_lnum_t *>(ms->row_index));
    const cs_lnum_t *__restrict__ d_col_id
      = (const cs_lnum_t *)cs_get_device_ptr_const_pf
                             (const_cast<cs_lnum_t *>(ms->col_id));
    const cs_real_t *__restrict__ d_val
      = (const cs_real_t *)cs_get_device_ptr_const_pf
                             (const_cast<cs_real_t *>(mc->val));

    unsigned int blocksize = 256;
    unsigned int gridsize
      = (unsigned int)ceil((double)ms->n_rows / blocksize);

    _mat_vect_p_l_csr_substract_diag<<<gridsize, blocksize, 0, _stream>>>
      (ms->n_rows, d_row_index, d_col_id, d_val,
       (const cs_real_t *)d_x, (cs_real_t *)d_y);

  }

  if (_stream == 0) {
    hipStreamSynchronize(0);
    CS_CUDA_CHECK(hipGetLastError());
  }
}

#endif /* defined(HAVE_CUSPARSE) */

/*----------------------------------------------------------------------------*/
/*!
 * \brief Matrix.vector product y = A.x with MSR matrix, scalar CUDA version.
 *
 * \param[in]   matrix        pointer to matrix structure
 * \param[in]   exclude_diag  exclude diagonal if true,
 * \param[in]   sync          synchronize ghost cells if true
 * \param[in]   d_x           multipliying vector values (on device)
 * \param[out]  d_y           resulting vector (on device)
 */
/*----------------------------------------------------------------------------*/

void
cs_matrix_spmv_cuda_msr(cs_matrix_t  *matrix,
                        bool          exclude_diag,
                        bool          sync,
                        cs_real_t     d_x[restrict],
                        cs_real_t     d_y[restrict])
{
  const cs_matrix_struct_dist_t *ms
    = (const cs_matrix_struct_dist_t *)matrix->structure;
  const cs_matrix_coeff_dist_t *mc
    = (const cs_matrix_coeff_dist_t *)matrix->coeffs;

  const cs_lnum_t *__restrict__ row_index
    = (const cs_lnum_t *)cs_get_device_ptr_const_pf
                           (const_cast<cs_lnum_t *>(ms->e.row_index));
  const cs_lnum_t *__restrict__ col_id
    = (const cs_lnum_t *)cs_get_device_ptr_const_pf
                          (const_cast<cs_lnum_t *>(ms->e.col_id));

  const cs_real_t *__restrict__ d_val
    = (const cs_real_t *)cs_get_device_ptr_const_pf
                           (const_cast<cs_real_t *>(mc->d_val));
  const cs_real_t *__restrict__ x_val
    = (const cs_real_t *)cs_get_device_ptr_const_pf
                           (const_cast<cs_real_t *>(mc->e_val));

  /* Ghost cell communication */

  if (sync) {
    cs_halo_state_t *hs = _pre_vector_multiply_sync_x_start(matrix, d_x);
    cs_halo_sync_wait(matrix->halo, d_x, hs);
  }

  /* Compute SpMV */

  unsigned int blocksize = 256;
  unsigned int gridsize
    = (unsigned int)ceil((double)ms->n_rows / blocksize);

  if (!exclude_diag)
    _mat_vect_p_l_msr<<<gridsize, blocksize, 0, _stream>>>
      (ms->n_rows, row_index, col_id, d_val, x_val, d_x, d_y);
  else
    _mat_vect_p_l_csr<<<gridsize, blocksize, 0, _stream>>>
      (ms->n_rows, row_index, col_id, x_val, d_x, d_y);

  if (_stream == 0) {
    hipStreamSynchronize(0);
    CS_CUDA_CHECK(hipGetLastError());
  }
}

#if defined(HAVE_CUSPARSE)

/*----------------------------------------------------------------------------*/
/*!
 * \brief Matrix.vector product y = A.x with MSR matrix, scalar cuSPARSE version.
 *
 * \param[in]   matrix        pointer to matrix structure
 * \param[in]   exclude_diag  exclude diagonal if true,
 * \param[in]   sync          synchronize ghost cells if true
 * \param[in]   d_x           multipliying vector values (on device)
 * \param[out]  d_y           resulting vector (on device)
 */
/*----------------------------------------------------------------------------*/

void
cs_matrix_spmv_cuda_msr_cusparse(cs_matrix_t  *matrix,
                                 bool          exclude_diag,
                                 bool          sync,
                                 cs_real_t     d_x[restrict],
                                 cs_real_t     d_y[restrict])
{
  cs_matrix_cusparse_map_t *csm
    = (cs_matrix_cusparse_map_t *)matrix->ext_lib_map;

  if (csm == NULL) {
    matrix->ext_lib_map = _set_cusparse_map(matrix);
    csm = (cs_matrix_cusparse_map_t *)matrix->ext_lib_map;
  }

  /* Ghost cell communication */

  if (sync) {
    cs_halo_state_t *hs = _pre_vector_multiply_sync_x_start(matrix,
                                                            (cs_real_t *)d_x);
    cs_halo_sync_wait(matrix->halo, (cs_real_t *)d_x, hs);
  }

  _update_cusparse_map(csm, matrix, d_x, d_y);

  cs_real_t alpha = 1.;
  cs_real_t beta = 0.;

  if (!exclude_diag) {

    const cs_matrix_struct_dist_t *ms
      = (const cs_matrix_struct_dist_t *)matrix->structure;
    const cs_matrix_coeff_dist_t *mc
      = (const cs_matrix_coeff_dist_t *)matrix->coeffs;
    const cs_real_t *__restrict__ d_val
      = (const cs_real_t *)cs_get_device_ptr_const_pf
                             (const_cast<cs_real_t *>(mc->d_val));

    unsigned int blocksize = 256;
    unsigned int gridsize
      = (unsigned int)ceil((double)ms->n_rows / blocksize);

    _mat_vect_p_l_msr_diag<<<gridsize, blocksize, 0, _stream>>>
      (ms->n_rows, d_val, (const cs_real_t *)d_x, (cs_real_t *)d_y);

    beta = 1.;

  }

  hipsparseSetStream(_handle, _stream);

  hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;

#if defined(HAVE_CUSPARSE_GENERIC_API)

#if CUSPARSE_VER_MAJOR >= 12
  hipsparseSpMVAlg_t spmv_alg_type = HIPSPARSE_SPMV_ALG_DEFAULT;
#else
  hipsparseSpMVAlg_t spmv_alg_type = HIPSPARSE_MV_ALG_DEFAULT;
#endif

  hipDataType val_dtype
    = (sizeof(cs_real_t) == 8) ? HIP_R_64F : HIP_R_32F;

  status = hipsparseSpMV(_handle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &alpha,
                        csm->matA,
                        csm->vecX,
                        &beta,
                        csm->vecY,
                        val_dtype,
                        spmv_alg_type,
                        csm->dBuffer);

  if (HIPSPARSE_STATUS_SUCCESS != status)
    bft_error(__FILE__, __LINE__, 0, _("%s: %s."),
              __func__, hipsparseGetErrorString(status));

#else

  if (sizeof(cs_real_t) == 8) {
    double _alpha = alpha;
    double _beta = beta;
    status = hipsparseDcsrmv(_handle,
                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            matrix->n_rows,
                            matrix->n_cols_ext,
                            csm->nnz,
                            &_alpha,
                            csm->descrA,
                            (const double *)csm->d_e_val,
                            (const int *)csm->d_row_index,
                            (const int *)csm->d_col_id,
                            (const double *)d_x,
                            &_beta,
                            (double *)d_y);
  }

  else if (sizeof(cs_real_t) == 4) {
    float _alpha = alpha;
    float _beta = beta;

    status = hipsparseScsrmv(_handle,
                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            matrix->n_rows,
                            matrix->n_cols_ext,
                            csm->nnz,
                            &_alpha,
                            csm->descrA,
                            (const float *)csm->d_e_val,
                            (const int *)csm->d_row_index,
                            (const int *)csm->d_col_id,
                            (const float *)d_x,
                            &_beta,
                            (float *)d_y);
  }

  if (HIPSPARSE_STATUS_SUCCESS != status)
    bft_error(__FILE__, __LINE__, 0, _("%s: cuSPARSE error %d."),
              __func__, (int)status);

#endif

  if (_stream == 0) {
    hipStreamSynchronize(0);
    CS_CUDA_CHECK(hipGetLastError());
  }
}

#endif /* defined(HAVE_CUSPARSE) */

/*----------------------------------------------------------------------------*/
/*!
 * \brief Matrix.vector product y = A.x with MSR matrix, block diagonal
 *        CUDA version.
 *
 * \param[in]   matrix        pointer to matrix structure
 * \param[in]   exclude_diag  exclude diagonal if true,
 * \param[in]   sync          synchronize ghost cells if true
 * \param[in]   d_x           multipliying vector values (on device)
 * \param[out]  d_y           resulting vector (on device)
 */
/*----------------------------------------------------------------------------*/

void
cs_matrix_spmv_cuda_msr_b(cs_matrix_t  *matrix,
                          bool          exclude_diag,
                          bool          sync,
                          cs_real_t     d_x[restrict],
                          cs_real_t     d_y[restrict])
{
  const cs_matrix_struct_dist_t *ms
    = (const cs_matrix_struct_dist_t *)matrix->structure;
  const cs_matrix_coeff_dist_t *mc
    = (const cs_matrix_coeff_dist_t *)matrix->coeffs;

  const cs_lnum_t *__restrict__ row_index
    = (const cs_lnum_t *)cs_get_device_ptr_const_pf
                           (const_cast<cs_lnum_t *>(ms->e.row_index));
  const cs_lnum_t *__restrict__ col_id
    = (const cs_lnum_t *)cs_get_device_ptr_const_pf
                           (const_cast<cs_lnum_t *>(ms->e.col_id));

  const cs_real_t *__restrict__ d_val
    = (const cs_real_t *)cs_get_device_ptr_const_pf
                           (const_cast<cs_real_t *>(mc->d_val));
  const cs_real_t *__restrict__ x_val
    = (const cs_real_t *)cs_get_device_ptr_const_pf
                           (const_cast<cs_real_t *>(mc->e_val));

  /* Ghost cell communication */

  if (sync) {
    cs_halo_state_t *hs = _pre_vector_multiply_sync_x_start(matrix, d_x);
    cs_halo_sync_wait(matrix->halo, d_x, hs);
  }

  /* Compute SpMV */

  unsigned int blocksize = 128;
  unsigned int gridsize
    = (unsigned int)ceil((double)ms->n_rows / blocksize);

  if (!exclude_diag) {

    if (matrix->db_size == 3)
      _b_3_3_mat_vect_p_l_msr<<<gridsize, blocksize, 0, _stream>>>
        (ms->n_rows, col_id, row_index, d_val, x_val, d_x, d_y);
    else if (matrix->db_size == 6)
      _b_mat_vect_p_l_msr<6><<<gridsize, blocksize, 0, _stream>>>
        (ms->n_rows, col_id, row_index, d_val, x_val, d_x, d_y);
    else if (matrix->db_size == 9)
      _b_mat_vect_p_l_msr<9><<<gridsize, blocksize, 0, _stream>>>
        (ms->n_rows, col_id, row_index, d_val, x_val, d_x, d_y);
    else
      bft_error(__FILE__, __LINE__, 0, _("%s: block size %d not implemented."),
                __func__, (int)matrix->db_size);

  }
  else {

    if (matrix->db_size == 3)
      _b_3_3_mat_vect_p_l_msr_exdiag<<<gridsize, blocksize, 0, _stream>>>
        (ms->n_rows, col_id, row_index, d_val, x_val, d_x, d_y);
    else if (matrix->db_size == 6)
      _b_mat_vect_p_l_msr_exdiag<6><<<gridsize, blocksize, 0, _stream>>>
        (ms->n_rows, col_id, row_index, d_val, x_val, d_x, d_y);
    else if (matrix->db_size == 9)
      _b_mat_vect_p_l_msr_exdiag<9><<<gridsize, blocksize, 0, _stream>>>
        (ms->n_rows, col_id, row_index, d_val, x_val, d_x, d_y);
    else
      bft_error(__FILE__, __LINE__, 0, _("%s: block size %d not implemented."),
                __func__, (int)matrix->db_size);

  }

  if (_stream == 0) {
    CS_CUDA_CHECK(hipStreamSynchronize(0));
    CS_CUDA_CHECK(hipGetLastError());
  }
}

#if defined(HAVE_CUSPARSE_GENERIC_API)

/*----------------------------------------------------------------------------*/
/*!
 * \brief Matrix.vector product y = A.x with MSR matrix, block diagonal
 *        cuSPARSE version.
 *
 * Remark: this functions is available with older cuSPARSE versions not
 *         providing the generic API, because they
 *         assume dense matrixes are always in column-major order, while
 *         row-major is needed with interleaved blocks.
 *
 * \param[in]   matrix        pointer to matrix structure
 * \param[in]   exclude_diag  exclude diagonal if true,
 * \param[in]   sync          synchronize ghost cells if true
 * \param[in]   d_x           multipliying vector values (on device)
 * \param[out]  d_y           resulting vector (on device)
 */
/*----------------------------------------------------------------------------*/

void
cs_matrix_spmv_cuda_msr_b_cusparse(cs_matrix_t  *matrix,
                                   bool          exclude_diag,
                                   bool          sync,
                                   cs_real_t     d_x[restrict],
                                   cs_real_t     d_y[restrict])
{
  cs_matrix_cusparse_map_t *csm
    = (cs_matrix_cusparse_map_t *)matrix->ext_lib_map;

  if (csm == NULL) {
    matrix->ext_lib_map = _set_cusparse_map(matrix);
    csm = (cs_matrix_cusparse_map_t *)matrix->ext_lib_map;
  }

  /* Ghost cell communication */

  if (sync) {
    cs_halo_state_t *hs = _pre_vector_multiply_sync_x_start(matrix,
                                                            (cs_real_t *)d_x);
    cs_halo_sync_wait(matrix->halo, (cs_real_t *)d_x, hs);
  }

  _update_cusparse_map_block_diag(csm, matrix, d_x, d_y);

  cs_real_t alpha = 1.;
  cs_real_t beta = 0.;

  if (!exclude_diag) {

    const cs_matrix_struct_dist_t *ms
      = (const cs_matrix_struct_dist_t *)matrix->structure;
    const cs_matrix_coeff_dist_t *mc
      = (const cs_matrix_coeff_dist_t *)matrix->coeffs;
    const cs_real_t *__restrict__ d_val
      = (const cs_real_t *)cs_get_device_ptr_const_pf
                             (const_cast<cs_real_t *>(mc->d_val));

    unsigned int blocksize = 128;
    unsigned int gridsize
      = (unsigned int)ceil((double)ms->n_rows / blocksize);

    if (matrix->db_size == 3)
      _b_3_3_spmv_diag<<<gridsize, blocksize, 0, _stream>>>
        (ms->n_rows, d_val, (const cs_real_t *)d_x, (cs_real_t *)d_y);
    else if (matrix->db_size == 6)
      _b_spmv_diag<6><<<gridsize, blocksize, 0, _stream>>>
        (ms->n_rows, d_val, (const cs_real_t *)d_x, (cs_real_t *)d_y);
    else if (matrix->db_size == 9)
      _b_spmv_diag<9><<<gridsize, blocksize, 0, _stream>>>
        (ms->n_rows, d_val, (const cs_real_t *)d_x, (cs_real_t *)d_y);
    else
      bft_error(__FILE__, __LINE__, 0, _("%s: block size %d not implemented."),
                __func__, (int)matrix->db_size);

    beta = 1.;

  }

  hipsparseSetStream(_handle, _stream);

  hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;

  hipDataType val_dtype
    = (sizeof(cs_real_t) == 8) ? HIP_R_64F : HIP_R_32F;

  status = hipsparseSpMM(_handle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &alpha,
                        csm->matA,
                        csm->matX,
                        &beta,
                        csm->matY,
                        val_dtype,
                        HIPSPARSE_SPMM_ALG_DEFAULT,
                        csm->dBuffer);

  if (HIPSPARSE_STATUS_SUCCESS != status)
    bft_error(__FILE__, __LINE__, 0, _("%s: %s."),
              __func__, hipsparseGetErrorString(status));

  if (_stream == 0) {
    hipStreamSynchronize(0);
    CS_CUDA_CHECK(hipGetLastError());
  }
}

#endif /* defined(HAVE_CUSPARSE_GENERIC_API) */

#if defined(HAVE_CUSPARSE)

/*----------------------------------------------------------------------------*/
/*!
 * \brief Matrix.vector product y = A.x with MSR matrix, block
 *        cuSPARSE version.
 *
 * Remmark: this functions is available with older cuSPARSE versions not
 *          providing the generic API, because they
 *          assume dense matrixes are always in column-major order, while
 *          row-major is needed with interleaved blocks.
 *
 * \param[in]   matrix        pointer to matrix structure
 * \param[in]   exclude_diag  exclude diagonal if true,
 * \param[in]   sync          synchronize ghost cells if true
 * \param[in]   d_x           multipliying vector values (on device)
 * \param[out]  d_y           resulting vector (on device)
 */
/*----------------------------------------------------------------------------*/

void
cs_matrix_spmv_cuda_msr_bb_cusparse(cs_matrix_t  *matrix,
                                    bool          exclude_diag,
                                    bool          sync,
                                    cs_real_t     d_x[restrict],
                                    cs_real_t     d_y[restrict])
{
  cs_matrix_cusparse_map_t *csm
    = (cs_matrix_cusparse_map_t *)matrix->ext_lib_map;

  if (csm == NULL) {
    matrix->ext_lib_map = _set_cusparse_map(matrix);
    csm = (cs_matrix_cusparse_map_t *)matrix->ext_lib_map;
  }

  /* Ghost cell communication */

  if (sync) {
    cs_halo_state_t *hs = _pre_vector_multiply_sync_x_start(matrix,
                                                            (cs_real_t *)d_x);
    cs_halo_sync_wait(matrix->halo, (cs_real_t *)d_x, hs);
  }

  /* no update_cusparse_map type function call here as only
     the non-generic API is available here */

  cs_real_t alpha = 1.;
  cs_real_t beta = 0.;

  if (!exclude_diag) {

    const cs_matrix_struct_dist_t *ms
      = (const cs_matrix_struct_dist_t *)matrix->structure;
    const cs_matrix_coeff_dist_t *mc
      = (const cs_matrix_coeff_dist_t *)matrix->coeffs;
    const cs_real_t *__restrict__ d_val
      = (const cs_real_t *)cs_get_device_ptr_const_pf
                             (const_cast<cs_real_t *>(mc->d_val));

    unsigned int blocksize = 128;
    unsigned int gridsize
      = (unsigned int)ceil((double)ms->n_rows / blocksize);

    if (matrix->db_size == 3)
      _b_3_3_spmv_diag<<<gridsize, blocksize, 0, _stream>>>
        (ms->n_rows, d_val, (const cs_real_t *)d_x, (cs_real_t *)d_y);
    else if (matrix->db_size == 6)
      _b_spmv_diag<6><<<gridsize, blocksize, 0, _stream>>>
        (ms->n_rows, d_val, (const cs_real_t *)d_x, (cs_real_t *)d_y);
    else if (matrix->db_size == 9)
      _b_spmv_diag<9><<<gridsize, blocksize, 0, _stream>>>
        (ms->n_rows, d_val, (const cs_real_t *)d_x, (cs_real_t *)d_y);
    else
      bft_error(__FILE__, __LINE__, 0, _("%s: block size %d not implemented."),
                __func__, (int)matrix->db_size);

    beta = 1.;

  }

  hipsparseSetStream(_handle, _stream);

  hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;

  if (sizeof(cs_real_t) == 8) {
    double _alpha = alpha;
    double _beta = beta;
    status = hipsparseDbsrmv(_handle,
                            HIPSPARSE_DIRECTION_ROW,
                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            matrix->n_rows,
                            matrix->n_cols_ext,
                            csm->nnz,
                            &_alpha,
                            csm->descrA,
                            (const double *)csm->d_e_val,
                            (const int *)csm->d_row_index,
                            (const int *)csm->d_col_id,
                            matrix->eb_size,
                            (const double *)d_x,
                            &_beta,
                            (double *)d_y);
  }

  else if (sizeof(cs_real_t) == 4) {
    float _alpha = alpha;
    float _beta = beta;

    status = hipsparseSbsrmv(_handle,
                            HIPSPARSE_DIRECTION_ROW,
                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            matrix->n_rows,
                            matrix->n_cols_ext,
                            csm->nnz,
                            &_alpha,
                            csm->descrA,
                            (const float *)csm->d_e_val,
                            (const int *)csm->d_row_index,
                            (const int *)csm->d_col_id,
                            matrix->eb_size,
                            (const float *)d_x,
                            &_beta,
                            (float *)d_y);
  }

  if (HIPSPARSE_STATUS_SUCCESS != status)
    bft_error(__FILE__, __LINE__, 0, _("%s: cuSPARSE error %d."),
              __func__, (int)status);

  if (_stream == 0) {
    hipStreamSynchronize(0);
    CS_CUDA_CHECK(hipGetLastError());
  }
}

#endif /* defined(HAVE_CUSPARSE) */

/*----------------------------------------------------------------------------*/

END_C_DECLS
