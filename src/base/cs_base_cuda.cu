#include "hip/hip_runtime.h"
/*============================================================================
 * Definitions, global variables, and base functions for CUDA
 *============================================================================*/

/*
  This file is part of code_saturne, a general-purpose CFD tool.

  Copyright (C) 1998-2023 EDF S.A.

  This program is free software; you can redistribute it and/or modify it under
  the terms of the GNU General Public License as published by the Free Software
  Foundation; either version 2 of the License, or (at your option) any later
  version.

  This program is distributed in the hope that it will be useful, but WITHOUT
  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
  FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
  details.

  You should have received a copy of the GNU General Public License along with
  this program; if not, write to the Free Software Foundation, Inc., 51 Franklin
  Street, Fifth Floor, Boston, MA 02110-1301, USA.
*/

/*----------------------------------------------------------------------------*/

#include "cs_defs.h"

/*----------------------------------------------------------------------------
 * Standard C library headers
 *----------------------------------------------------------------------------*/

/*----------------------------------------------------------------------------
 * Local headers
 *----------------------------------------------------------------------------*/

#include "assert.h"
#include "bft_error.h"
#include "bft_mem.h"
#include "bft_printf.h"

#include "cs_base.h"
#include "cs_log.h"

/*----------------------------------------------------------------------------
 *  Header for the current file
 *----------------------------------------------------------------------------*/

#include "cs_base_cuda.h"

/*----------------------------------------------------------------------------*/

BEGIN_C_DECLS

/*! \cond DOXYGEN_SHOULD_SKIP_THIS */

/*============================================================================
 * Local Macro Definitions
 *============================================================================*/

/*============================================================================
 * Local Type Definitions
 *============================================================================*/

/*============================================================================
 *  Global variables
 *============================================================================*/

/* Keep track of active device id; usually queried dynamically, but
   saving the value in this variable can be useful when debugging */

int  cs_glob_cuda_device_id = -1;

/* Other device parameters */

int  cs_glob_cuda_max_threads_per_block = -1;
int  cs_glob_cuda_max_block_size = -1;
int  cs_glob_cuda_max_blocks = -1;
int  cs_glob_cuda_n_mp = -1;

/*============================================================================
 * Private function definitions
 *============================================================================*/

/*============================================================================
 * Semi-private function prototypes
 *
 * The following functions are intended to be used by the common
 * host-device memory management functions from cs_base_accel.c, and
 * not directly by the user.
 *============================================================================*/

/*----------------------------------------------------------------------------*/
/*!
 * \brief Allocate n bytes of CUDA device memory.
 *
 * This function simply wraps hipMalloc, which could probably be
 * directly called from C or C++, but whose use in such manner is not
 * well documented, and whose declaration in hip/hip_runtime.h requires
 * support of function attributes by compiler.
 *
 * A safety check is added.
 *
 * \param [in]  n          element size
 * \param [in]  var_name   allocated variable name string
 * \param [in]  file_name  name of calling source file
 * \param [in]  line_num   line number in calling source file
 *
 * \returns pointer to allocated memory.
 */
/*----------------------------------------------------------------------------*/

void *
cs_cuda_mem_malloc_device(size_t        n,
                          const char   *var_name,
                          const char   *file_name,
                          int           line_num)
{
  void *ptr = NULL;

  CS_CUDA_CHECK_CALL(hipMalloc(&ptr, n), file_name, line_num);

  return ptr;
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Allocate n bytes of host memory using CUDA.
 *
 * This function simply wraps hipHostMalloc, which could probably be
 * directly called from C or C++, but whose use in such manner is not
 * well documented, and whose declaration in hip/hip_runtime.h requires
 * support of function attributes by compiler.
 *
 * A safety check is added.
 *
 * \param [in]  n          element size
 * \param [in]  var_name   allocated variable name string
 * \param [in]  file_name  name of calling source file
 * \param [in]  line_num   line number in calling source file
 *
 * \returns pointer to allocated memory.
 */
/*----------------------------------------------------------------------------*/

void *
cs_cuda_mem_malloc_host(size_t        n,
                        const char   *var_name,
                        const char   *file_name,
                        int           line_num)
{
  void *ptr = NULL;

  CS_CUDA_CHECK_CALL(hipHostMalloc(&ptr, n), file_name, line_num);

  return ptr;
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Allocate n bytes of CUDA managed memory.
 *
 * This function simply wraps hipMallocManaged, which could probably be
 * directly called from C or C++, but whose use in such manner is not
 * well documented, and whose declaration in hip/hip_runtime.h requires
 * support of function attributes by compiler.
 *
 * A safety check is added.
 *
 * \param [in]  n          element size
 * \param [in]  var_name   allocated variable name string
 * \param [in]  file_name  name of calling source file
 * \param [in]  line_num   line number in calling source file
 *
 * \returns pointer to allocated memory.
 */
/*----------------------------------------------------------------------------*/

void *
cs_cuda_mem_malloc_managed(size_t        n,
                           const char   *var_name,
                           const char   *file_name,
                           int           line_num)
{
  void *ptr = NULL;

  CS_CUDA_CHECK_CALL(hipMallocManaged(&ptr, n), file_name, line_num);

#if 0
  CS_CUDA_CHECK_CALL(hipMemPrefetchAsync (*pointer, size, hipCpuDeviceId, 0),
                     file_name, line_num);
  CS_CUDA_CHECK_CALL(hipDeviceSynchronize(), file_name, line_num);
#endif

  return ptr;
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Free CUDA memory associated with a given pointer.
 *
 * This function simply wraps hipFree, which could probably be
 * directly called from C or C++, but whose use in such manner is not
 * well documented, and whose declaration in hip/hip_runtime.h requires
 * support of function attributes by compiler.
 *
 * A safety check is added.
 *
 * \param [in]  p          pointer to device memory
 * \param [in]  var_name   allocated variable name string
 * \param [in]  file_name  name of calling source file
 * \param [in]  line_num   line number in calling source file
 *
 * \returns pointer to allocated memory.
 */
/*----------------------------------------------------------------------------*/

void
cs_cuda_mem_free(void         *p,
                 const char   *var_name,
                 const char   *file_name,
                 int           line_num)
{
  CS_CUDA_CHECK_CALL(hipFree(p), file_name, line_num);

#if 0
  CS_CUDA_CHECK_CALL((hipDeviceSynchronize(), file_name, line_num));
#endif
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Free CUDA-allocated host memory associated with a given pointer.
 *
 * This function simply wraps hipHostFree, which could probably be
 * directly called from C or C++, but whose use in such manner is not
 * well documented, and whose declaration in hip/hip_runtime.h requires
 * support of function attributes by compiler.
 *
 * A safety check is added.
 *
 * \param [in]  p          pointer to device memory
 * \param [in]  var_name   allocated variable name string
 * \param [in]  file_name  name of calling source file
 * \param [in]  line_num   line number in calling source file
 *
 * \returns pointer to allocated memory.
 */
/*----------------------------------------------------------------------------*/

void
cs_cuda_mem_free_host(void         *p,
                      const char   *var_name,
                      const char   *file_name,
                      int           line_num)
{
  CS_CUDA_CHECK_CALL(hipHostFree(p), file_name, line_num);

#if 0
  CS_CUDA_CHECK_CALL((hipDeviceSynchronize(), file_name, line_num));
#endif
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Copy data from host to device.
 *
 * This is simply a wrapper over hipMemcpy.
 *
 * A safety check is added.
 *
 * \param [out]  dst   pointer to destination data
 * \param [in]   src   pointer to source data
 * \param [in]   size  size of data to copy
 */
/*----------------------------------------------------------------------------*/

void
cs_cuda_copy_h2d(void        *dst,
                 const void  *src,
                 size_t       size)
{
  CS_CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Copy data from host to device, possibly returning on the host
 *        before the copy is finished.
 *
 * This is simply a wrapper over hipMemcpyAsync.
 *
 * A safety check is added.
 *
 * \param [out]  dst   pointer to destination data
 * \param [in]   src   pointer to source data
 * \param [in]   size  size of data to copy
 *
 * \returns pointer to allocated memory.
 */
/*----------------------------------------------------------------------------*/

void
cs_cuda_copy_h2d_async(void        *dst,
                       const void  *src,
                       size_t       size)
{
  CS_CUDA_CHECK(hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice));
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Copy data from device to host.
 *
 * This is simply a wrapper over hipMemcpy.
 *
 * A safety check is added.
 *
 * \param [out]  dst   pointer to destination data
 * \param [in]   src   pointer to source data
 * \param [in]   size  size of data to copy
 *
 * \returns pointer to allocated memory.
 */
/*----------------------------------------------------------------------------*/

void
_cs_cuda_copy_d2h(void        *dst,
                 const void  *src,
                 size_t       size,
                 const char* filename,
                 long line)
{
  CS_CUDA_CHECK_CALL(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost), filename, line);
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Copy data from host to device.
 *
 * This is simply a wrapper over hipMemcpy.
 *
 * A safety check is added.
 *
 * \param [out]  dst   pointer to destination data
 * \param [in]   src   pointer to source data
 * \param [in]   size  size of data to copy
 *
 * \returns pointer to allocated memory.
 */
/*----------------------------------------------------------------------------*/

void
cs_cuda_copy_d2h_async(void        *dst,
                       const void  *src,
                       size_t       size)
{
  CS_CUDA_CHECK(hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost));
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Copy data from host to device.
 *
 * This is simply a wrapper over hipMemcpy.
 *
 * A safety check is added.
 *
 * \param [out]  dst   pointer to destination data
 * \param [in]   src   pointer to source data
 * \param [in]   size  size of data to copy
 *
 * \returns pointer to allocated memory.
 */
/*----------------------------------------------------------------------------*/

void
cs_cuda_prefetch_h2d(void    *dst,
                     size_t   size)
{
  CS_CUDA_CHECK(hipMemPrefetchAsync(dst, size, cs_glob_cuda_device_id, 0));
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Copy data from device to host.
 *
 * This is simply a wrapper over hipMemcpy.
 *
 * A safety check is added.
 *
 * \param [out]  dst   pointer to destination data
 * \param [in]   src   pointer to source data
 * \param [in]   size  size of data to copy
 *
 * \returns pointer to allocated memory.
 */
/*----------------------------------------------------------------------------*/

void
cs_cuda_prefetch_d2h(void    *dst,
                     size_t   size)
{
  CS_CUDA_CHECK(hipMemPrefetchAsync(dst, size, hipCpuDeviceId, 0));
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Copy data from device to device.
 *
 * This is simply a wrapper over hipMemcpy.
 *
 * A safety check is added.
 *
 * \param [out]  dst   pointer to destination data
 * \param [in]   src   pointer to source data
 * \param [in]   size  size of data to copy
 */
/*----------------------------------------------------------------------------*/

void
cs_cuda_copy_d2d(void        *dst,
                 const void  *src,
                 size_t       size)
{
  CS_CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice));
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Get host pointer for a managed or device pointer.
 *
 * This function can be called with a pointer inside an allocated block of
 * memory, so is not retricted to values returned by CS_ALLOC_HD.
 *
 * This makes it possible to check whether a pointer to an array inside
 * a larger array is shared or accessible from the device only
 * (for example when grouping allocations).
 *
 * \param [in]   ptr   pointer to device data
 *
 * \return  pointer to host data if shared or mapped at the CUDA level,
 *          NULL otherwise.
 */
/*----------------------------------------------------------------------------*/

void *
cs_cuda_get_host_ptr(const void  *ptr)
{
  hipPointerAttribute_t attributes;

  void *host_ptr = NULL;
  int retcode = hipPointerGetAttributes(&attributes, ptr);

  if (retcode == hipSuccess) {
    if (ptr != attributes.devicePointer)
      bft_error(__FILE__, __LINE__, 0,
                _("%s: %p does not seem to be a managed or device pointer."),
                __func__, ptr);

    host_ptr = attributes.hostPointer;
  }

  return host_ptr;
}

/*! (DOXYGEN_SHOULD_SKIP_THIS) \endcond */

/*============================================================================
 * Public function definitions
 *============================================================================*/

/*----------------------------------------------------------------------------*/
/*!
 * \brief  Log information on available CUDA devices.
 *
 * \param[in]  log_id  id of log file in which to print information
 */
/*----------------------------------------------------------------------------*/

void
cs_base_cuda_device_info(cs_log_t  log_id)
{
  int n_devices = 0;

  hipError_t retval = hipGetDeviceCount(&n_devices);

  if (retval == hipErrorNoDevice)
    cs_log_printf(log_id,
                  _("  CUDA device:         none available\n"));
  else if (retval)
    cs_log_printf(log_id,
                  _("  CUDA device:         %s\n"),
                  hipGetErrorString(retval));

  char buffer[256] = "";

  for (int i = 0; i < n_devices; i++) {
    struct hipDeviceProp_t prop;
    CS_CUDA_CHECK(hipGetDeviceProperties(&prop, i));
    unsigned long long mem = prop.totalGlobalMem / 1000000;

    cs_log_printf
      (log_id,
       _("  CUDA device %d:       %s\n"),
       i, prop.name);

    if (strncmp(prop.name, buffer, 255) != 0) {
      cs_log_printf
        (log_id,
         _("                       Compute capability: %d.%d\n"
           "                       Memory: %llu %s\n"
           "                       Multiprocessors: %d\n"
           "                       Integrated: %d\n"
           "                       Unified addressing: %d\n"),
         prop.major, prop.minor,
         mem, _("MB"),
         prop.multiProcessorCount,
         prop.integrated,
         prop.unifiedAddressing);

#if (CUDART_VERSION >= 11000)
      cs_log_printf
        (log_id,
         _("                       Use host's page tables: %d\n"),
         prop.pageableMemoryAccessUsesHostPageTables);
#endif
    }

    strncpy(buffer, prop.name, 255);
    buffer[255] = '\0';
  }
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief  Log information on available CUDA version.
 *
 * \param[in]  log_id  id of log file in which to print information
 */
/*----------------------------------------------------------------------------*/

void
cs_base_cuda_version_info(cs_log_t  log_id)
{
  int runtime_version = -1, driver_version = -1;

  if (hipDriverGetVersion(&driver_version) == hipSuccess)
    cs_log_printf(log_id,
                  "  %s%d\n", _("CUDA driver:         "), driver_version);
  if (hipRuntimeGetVersion(&runtime_version) == hipSuccess)
    cs_log_printf(log_id,
                  "  %s%d\n", _("CUDA runtime:        "), runtime_version);
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief  Log information on CUDA compiler.
 *
 * \param[in]  log_id  id of log file in which to print information
 */
/*----------------------------------------------------------------------------*/

void
cs_base_cuda_compiler_info(cs_log_t  log_id)
{
  cs_log_printf(log_id,
                "    %s%d.%d.%d\n", _("CUDA compiler:     "),
                __CUDACC_VER_MAJOR__,
                __CUDACC_VER_MINOR__,
                __CUDACC_VER_BUILD__);
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Set CUDA device based on MPI rank and number of devices.
 *
 * \param[in]  comm            associated MPI communicator
 * \param[in]  ranks_per_node  number of ranks per node (min and max)
 *
 * \return  selected device id, or -1 if no usable device is available
 */
/*----------------------------------------------------------------------------*/

int
cs_base_cuda_select_default_device(void)
{
  int device_id = 0, n_devices = 0;

  hipError_t ret_code = hipGetDeviceCount(&n_devices);

  if (ret_code == hipErrorNoDevice)
    return -1;

  if (hipSuccess != ret_code) {
    cs_base_warn(__FILE__, __LINE__);
    bft_printf("[CUDA error] %d: %s\n  running: %s\n  in: %s\n",
               ret_code, ::hipGetErrorString(ret_code),
               "hipGetDeviceCount", __func__);
    return -1;
  }

  if (cs_glob_rank_id > -1 && n_devices > 1) {

    device_id = cs_glob_node_rank_id*n_devices / cs_glob_node_n_ranks;

    assert(device_id > -1 && device_id < n_devices);

  }

  ret_code = hipSetDevice(device_id);

  if (hipSuccess != ret_code) {
    cs_base_warn(__FILE__, __LINE__);
    bft_printf("[CUDA error] %d: %s\n  running: %s\n  in: %s\n",
               ret_code, ::hipGetErrorString(ret_code),
               "hipSetDevice", __func__);
    return -1;
  }

  cs_glob_cuda_device_id = device_id;

  /* Also query some device properties */

  struct hipDeviceProp_t prop;
  CS_CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
  cs_glob_cuda_max_threads_per_block = prop.maxThreadsPerBlock;
  cs_glob_cuda_max_block_size = prop.maxThreadsPerMultiProcessor;
  cs_glob_cuda_max_blocks
    =   prop.multiProcessorCount
      * (prop.maxThreadsPerMultiProcessor / prop.maxThreadsPerBlock);
  cs_glob_cuda_n_mp = prop.multiProcessorCount;

  return device_id;
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Return currently selected CUDA devices.
 *
 * \return  selected device id, or -1 if no usable device is available
 */
/*----------------------------------------------------------------------------*/

int
cs_base_cuda_get_device(void)
{
  int device_id = -1, n_devices = 0;

  hipError_t ret_code = hipGetDeviceCount(&n_devices);

  if (hipSuccess == ret_code)
    ret_code = hipGetDevice(&device_id);

  if (hipSuccess != ret_code)
    device_id = -1;

  return device_id;
}

/*----------------------------------------------------------------------------*/

END_C_DECLS
